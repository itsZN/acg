#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );


RT_PROGRAM void pinhole_camera()
{

  size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

  // Subpixel jitter: send the ray through a different position inside the pixel each time,
  // to provide antialiasing.
  float2 subpixel_jitter = frame == 0 ? make_float2(0.0f, 0.0f) : make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f);

  float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  float4 acc_val = accum_buffer[launch_index];
  if( frame > 0 ) {
    acc_val = lerp( acc_val, make_float4( prd.result, 0.f), 1.0f / static_cast<float>( frame+1 ) );
  } else {
    acc_val = make_float4(prd.result, 0.f);
  }
  // CHALLENGES
  // 1. We cannot get the normal and the light vectors of the first hit with this API,
  //    which are crucial to determine the intensity of the light. Instead we approximate
  //    this using RGB to intensity conversion.

#if 0
  /// perform the conversion to luminosity
  {
    float3 RGBtoY; RGBtoY.x = 0.2126; RGBtoY.y = 0.7152; RGBtoY.z = 0.0722;
    float3 mult1; mult1.x = 1.00; mult1.y = 1.00; mult1.z = 1.00;
    float3 mult2; mult2.x = 0.90; mult2.y = 0.90; mult2.z = 0.90;
    float3 mult3; mult3.x = 0.80; mult3.y = 0.80; mult3.z = 0.80;
    float3 mult4; mult4.x = 0.60; mult4.y = 0.60; mult4.z = 0.60;

    float3 color = make_float3(acc_val);
    // convert from RGB to Y, note that color.x,y,z are in [0, 1)
    float intensity = dot(RGBtoY, color);
    if (intensity > 0.95)
        color = mult1 * color;
    else if (intensity > 0.5)
        color = mult2 * color;
    else if (intensity > 0.05)
        color = mult3 * color;
    else
        color = mult4 * color;
    output_buffer[launch_index] = make_color( color );
  }
  /// end conversion to luminosity
#endif

  float3 color = make_float3( acc_val );
  color.x = ceil(color.x);
  color.y = ceil(color.y);
  color.z = ceil(color.z);

  output_buffer[launch_index] = make_color( color );
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}
