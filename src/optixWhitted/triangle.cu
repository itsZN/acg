#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3,  x, , );
rtDeclareVariable(float3,  y, , );
rtDeclareVariable(float3,  z, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void robust_intersect(int primIdx)
{
    // check if we intersect the plane of the triangle
    float3 n = normalize(cross(y - x, z - x));
    float d = dot(n, x);

    float t = (d - dot(n, ray.origin)) / dot(n, ray.direction);
    if (t < 0.00001)
        return;
    // check that point is within triangle
    float3 p = ray.direction * t + ray.origin;
    float3 tmp[3] = { x, y, z };
    for (int i = 0; i < 3; ++i) {
        float3 v1 = tmp[i] - p;
        float3 v2 = tmp[(i+1) % 3] - p;
        float3 n = normalize(cross(v2, v1));
        float d = dot(-ray.origin, n);
        if (dot(p,n) + d < 0)
            return;
    }
    if (rtPotentialIntersection(t)) {
        geometric_normal = shading_normal = n;
        rtReportIntersection(0);
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    result[0] = x.x;
    result[0] = min(result[0], y.x);
    result[0] = min(result[0], z.x);

    result[1] = x.x;
    result[1] = max(result[1], y.x);
    result[1] = max(result[1], z.x);

    result[2] = x.y;
    result[2] = min(result[2], y.y);
    result[2] = min(result[2], z.y);

    result[3] = x.y;
    result[3] = max(result[3], y.y);
    result[3] = max(result[3], z.y);

    result[4] = x.z;
    result[4] = min(result[4], y.z);
    result[4] = min(result[4], z.z);

    result[5] = x.z;
    result[5] = max(result[5], y.z);
    result[5] = max(result[5], z.z);
}
