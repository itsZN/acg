#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3,  x, , );
rtDeclareVariable(float3,  y, , );
rtDeclareVariable(float3,  z, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void robust_intersect(int primIdx)
{
    // check if we intersect the plane of the triangle
    float3 n = normalize(cross(y - x, z - x));
    float d = dot(n, x);

    float t = (d - dot(n, ray.origin)) / dot(n, ray.direction);
    if (t < 0.00001)
        return;
    rtPotentialIntersection(t);

    // check that point is within triangle
    float3 p = ray.direction * t + ray.origin;
    float3 tmp[3] = { x, y, z };
    for (int i = 0; i < 3; ++i) {
        float3 v1 = tmp[i] - p;
        float3 v2 = tmp[(i+1) % 3] - p;
        float3 n = normalize(cross(v2, v1));
        float d = dot(-ray.origin, n);
        if (dot(p,n) + d < 0)
            return;
    }
    rtReportIntersection(0);
}


RT_PROGRAM void bounds (int, float result[6])
{
    result[0] = x.x;
    result[0] = min(result[0], y.x);
    result[0] = min(result[0], z.x);

    result[1] = x.x;
    result[1] = max(result[1], y.x);
    result[1] = max(result[1], z.x);

    result[2] = x.y;
    result[2] = min(result[2], y.y);
    result[2] = min(result[2], z.y);

    result[3] = x.y;
    result[3] = max(result[3], y.y);
    result[3] = max(result[3], z.y);

    result[4] = x.z;
    result[4] = min(result[4], y.z);
    result[4] = min(result[4], z.z);

    result[5] = x.z;
    result[5] = max(result[5], y.z);
    result[5] = max(result[5], z.z);
}
